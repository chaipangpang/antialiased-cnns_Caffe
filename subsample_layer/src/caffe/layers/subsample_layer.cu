#include "hip/hip_runtime.h"
#include <algorithm>
#include <cfloat>
#include <vector>

#include "caffe/layers/subsample_layer.hpp"
#include "caffe/util/math_functions.hpp"

namespace caffe {

template <typename Dtype>
__global__ void SubSampleForward(const int nthreads,
    const Dtype* const bottom_data, const int num, const int channels,
    const int height, const int width, const int subsample_height,
	const int subsample_width, const int kernel_h, const int kernel_w,
    const int stride_h, const int stride_w, const int pad_h, const int pad_w,
    Dtype* const top_data) {
  CUDA_KERNEL_LOOP(index, nthreads) {
	  const int sw = index % subsample_width;
	  const int sh = (index / subsample_width) % subsample_height;
	  const int c = (index / subsample_width / subsample_height) % channels;
	  const int n = index / subsample_width / subsample_height / channels;
    int hstart = sh * stride_h - pad_h;
    int wstart = sw * stride_w - pad_w;
    hstart = max(hstart, 0);
    wstart = max(wstart, 0);
    const Dtype* const bottom_slice = bottom_data + (n * channels + c) * height * width;
	top_data[index] = bottom_slice[hstart * width + wstart];
  }
}

template <typename Dtype>
void SubsampleLayer<Dtype>::Forward_gpu(const vector<Blob<Dtype>*>& bottom,
      const vector<Blob<Dtype>*>& top) {
  const Dtype* bottom_data = bottom[0]->gpu_data();
  Dtype* top_data = top[0]->mutable_gpu_data();
  int count = top[0]->count();
   SubSampleForward<Dtype><<<CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS>>>(
        count, bottom_data, bottom[0]->num(), channels_,
        height_, width_, subsampled_height_, subsampled_width_, kernel_h_,
        kernel_w_, stride_h_, stride_w_, pad_h_, pad_w_, top_data);
}

template <typename Dtype>
__global__ void SubSampleBackward(const int nthreads, const Dtype* const top_diff,
    const int num, const int channels, const int height,
	const int width, const int subsample_height, const int subsample_width,
    const int kernel_h, const int kernel_w, const int stride_h,
    const int stride_w, const int pad_h, const int pad_w,
    Dtype* const bottom_diff) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    // find out the local index
    // find out the local offset
    const int w = index % width + pad_w;
    const int h = (index / width) % height + pad_h;
    const int c = (index / width / height) % channels;
    const int n = index / width / height / channels;
    const int shstart = (h < kernel_h) ? 0 : (h - kernel_h) / stride_h + 1;
    const int swstart = (w < kernel_w) ? 0 : (w - kernel_w) / stride_w + 1;
    const Dtype* const top_diff_slice = top_diff + (n * channels + c) * subsample_height * subsample_width;
	bottom_diff[index] = top_diff_slice[shstart * subsample_width + swstart] ;

  }
}


template <typename Dtype>
void SubsampleLayer<Dtype>::Backward_gpu(const vector<Blob<Dtype>*>& top,
      const vector<bool>& propagate_down, const vector<Blob<Dtype>*>& bottom) {
  if (!propagate_down[0]) {
    return;
  }
  const Dtype* top_diff = top[0]->gpu_diff();
  Dtype* bottom_diff = bottom[0]->mutable_gpu_diff();
  const int count = bottom[0]->count();
  caffe_gpu_set(count, Dtype(0.), bottom_diff);

  SubSampleBackward<Dtype> << <CAFFE_GET_BLOCKS(count), CAFFE_CUDA_NUM_THREADS >> >(
          count, top_diff, top[0]->num(), channels_,
		  height_, width_, subsampled_height_, subsampled_width_, kernel_h_,
          kernel_w_, stride_h_, stride_w_, pad_h_, pad_w_, bottom_diff);
   
  CUDA_POST_KERNEL_CHECK;
}

INSTANTIATE_LAYER_GPU_FUNCS(SubsampleLayer);

}  // namespace caffe
